#include <multi_gpu_policy.h>
#include <hip/hip_runtime.h>



hipError_t get_current_device(int * i_dev, int * is_primary, int * is_clean) {

    hipError_t cuda_err;
    hipError_t ierr;

    // GET current device bound to this thread (this will work for _both_ the
    // cuda runtim API and the cuda driver)
    cuda_err = hipGetDevice(i_dev);
    if (cuda_err != hipSuccess) {
        * i_dev      = -1;
        * is_primary = -1;
        * is_clean   = -1;
        return hipErrorInvalidContext;
    }

    // GET the state of the primary context
    unsigned int flags;
    int active;
    hipDevicePrimaryCtxGetState(* i_dev, & flags, & active);
    if (active == 1){
        * is_primary = 1;
        * is_clean   = 0;
        return hipSuccess;
    }

    // The PRIMARY CONTEXT could be the only context on the device bound to
    // this thread -- and just not be active because nothing has been called
    // it, or another context is the current context

    hipDevice_t device;

    ierr = hipCtxGetDevice(& device);
    if (ierr == hipErrorInvalidContext) {
        * is_primary = 1;
        * is_clean   = 1;
    } else if (ierr != hipSuccess) {
        return ierr;
    }

    // There is defintely a device bound to this thread -- so there MUST be a
    // current context. The only thing to figure out now is if this context is
    // the primary context (which just hasn't been used yet), or another.

    hipCtx_t context;

    ierr = hipCtxGetCurrent(& context);
    if (ierr != hipSuccess)
        return ierr;

    hipCtx_t primary_context;
    // NOTE: this will make the primary context active -- we need to release it
    // again below:
    ierr = hipDevicePrimaryCtxRetain(& primary_context, device);
    if (ierr != hipSuccess)
        return ierr;
    // Restore the pre-existing context by de-activating the primary context
    // that cudaPrimaryCtxRetain activated
    ierr = hipDevicePrimaryCtxRelease(device);
    if (ierr != hipSuccess)
        return ierr;

    if (primary_context == context) {
        * is_primary = 1;
        * is_clean   = 0;
        return hipSuccess;
    }

    * is_primary = 0;
    * is_clean   = 0;
    return hipSuccess;
}
